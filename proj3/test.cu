#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <cmath>
#include <cassert>
#include <unistd.h>
#include <fcntl.h>
#include <cstdio>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>
#include <iostream>
#include <iomanip>

using namespace std;

void swap(int &i) {
    // Some of the & are superfluous.
    i =
     (0xff&(i >> 24)) |
     (0xff00&(i >> 8)) |
     (0xff0000&(i << 8)) |
     (0xff000000&(i << 24));
}

int read_int(int fd) {
    int rv;
    int i;
    rv = read(fd, &i, 4); assert(rv == 4);
    swap(i);
    return i;
}

void
output_pgm(const std::string &fn, const float (&img)[28][28]) {

    std::ofstream ofs(fn, std::fstream::out|std::fstream::trunc);

    ofs << "P2\n";
    ofs << "28 28\n";
    ofs << "255\n";
    for (int i = 0; i < 28; i++) {
        for (int j = 0; j < 28; j++) {
            if (j > 0) {
                ofs << " ";
            }
            ofs << 255 - int(std::round(127.5*(img[i][j] + 1)));
        }
        ofs << "\n";
    }
}


template <int N>
void
read_mnist_images(const std::string &fn, float (&imgs)[N][28][28]) {

    int rv;

    int fd;
    fd = open(fn.c_str(), O_RDONLY);
    assert(fd >= 0);

    int magic = read_int(fd);
    assert(magic == 0x803);

    int n_images = read_int(fd);
    assert(n_images == N);

    int n_rows = read_int(fd);
    assert(n_rows == 28);

    int n_cols = read_int(fd);
    assert(n_cols == 28);

    for (int i = 0; i < N; i++) {
        unsigned char tmp[28][28];
        rv = read(fd, tmp, 28*28); assert(rv == 28*28);
        for (int r = 0; r < 28; r++) {
            for (int c = 0; c < 28; c++) {
                // Make go from -1 to 1.
                imgs[i][r][c] = double(tmp[r][c])/127.5 - 1;
            }
        }
    }

    rv = close(fd); assert(rv == 0);
}

template <int N>
void
read_mnist_labels(const std::string &fn, unsigned char (&labels)[N]) {

    int rv;

    int fd;
    fd = open(fn.c_str(), O_RDONLY);
    assert(fd >= 0);

    int magic = read_int(fd);
    assert(magic == 0x801);

    int n_labels = read_int(fd);
    assert(n_labels == N);

    rv = read(fd, labels, N); assert(rv == N);
    for (int i = 0; i < N; i++) {
        assert(labels[i] >= 0 && labels[i] <= 9);
    }

    rv = close(fd); assert(rv == 0);
}

int main(void)
{
    // allocate three device_vectors with 10 elements
    string imagename = "/data/home/avanroi1/cs580f/cs580/proj3/training/train-images-idx3-ubyte";
    string labelname = "/data/home/avanroi1/cs580f/cs580/proj3/training/train-labels-idx1-ubyte";

    static float training_images[60'000][28][28];
    read_mnist_images(imagename, training_images);
    //output_pgm("img0.pgm", training_images[0]);
    //output_pgm("img59999.pgm", training_images[59999]);

    static unsigned char training_labels[60'000];
    read_mnist_labels(labelname, training_labels);
    assert(training_labels[0] == 5);
    assert(training_labels[59'999] == 8);

    thrust::device_vector<int> X(10);
    thrust::device_vector<int> Y(10);
    thrust::device_vector<int> Z(10);
    //just tryna scrap
    // initialize X to 0,1,2,3, ....
    thrust::sequence(X.begin(), X.end());

    // compute Y = -X
    thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());
    thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<int>(std::cout, "\n"));

    // fill Z with twos
    thrust::fill(Z.begin(), Z.end(), 2);

    // compute Y = X mod 2
    thrust::transform(X.begin(), X.end(), Z.begin(), Y.begin(), thrust::modulus<int>());

    // replace all the ones in Y with tens
    thrust::replace(Y.begin(), Y.end(), 1, 10);

    // print Y
    thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<int>(std::cout, "\n"));

   
    return 0;    
}